#include "hip/hip_runtime.h"
/*
	CISC372 Assignment 4: Lost in space 
	Contributors: Patrick Harris, Robert Reardon
	File: cuda.c
*/

#include <stdlib.h>
#include <math.h>
#include "vector.h"
#include "config.h"
#include <hip/hip_runtime.h>

__global__ void pairwise_acceleration(int num_entities, double* pos_x, double* pos_y, double* pos_z, double* mass, double* accels)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < num_entities && j < num_entities && i != j)
    {
        double distance_x = pos_x[i] - pos_x[j];
        double distance_y = pos_y[i] - pos_y[j];
        double distance_z = pos_z[i] - pos_z[j];
        double magnitude_sq = distance_x * distance_x + distance_y * distance_y + distance_z * distance_z;
        double magnitude = sqrt(magnitude_sq);
        double accelmag = -1 * GRAV_CONSTANT * mass[j] / magnitude_sq;
        accels[i * num_entities + j] = accelmag * distance_x / magnitude;
        accels[i * num_entities + j + num_entities * num_entities] = accelmag * distance_y / magnitude;
        accels[i * num_entities + j + 2 * num_entities * num_entities] = accelmag * distance_z / magnitude;
    }
}

__global__ void row_summation(int num_entities, double* accels, double* vel_x, double* vel_y, double* vel_z, double* pos_x, double* pos_y, double* pos_z)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_entities)
    {
        double accel_sum_x = 0;
        double accel_sum_y = 0;
        double accel_sum_z = 0;
        for (int j = 0; j < num_entities; j++)
        {
            accel_sum_x += accels[i * num_entities + j];
            accel_sum_y += accels[i * num_entities + j + num_entities * num_entities];
            accel_sum_z += accels[i * num_entities + j + 2 * num_entities * num_entities];
        }
        vel_x[i] += accel_sum_x * INTERVAL;
        vel_y[i] += accel_sum_y * INTERVAL;
        vel_z[i] += accel_sum_z * INTERVAL;
        pos_x[i] += vel_x[i] * INTERVAL;
        pos_y[i] += vel_y[i] * INTERVAL;
        pos_z[i] += vel_z[i] * INTERVAL;
    }
}

void compute()
{
    double* d_pos_x;
    double* d_pos_y;
    double* d_pos_z;
    double* d_mass;
    double* d_accels;
    double* d_vel_x;
    double* d_vel_y;
    double* d_vel_z;
    int size = NUMENTITIES * sizeof(double);

    // Allocate device memory
    hipMalloc((void**)&d_pos_x, size);
    hipMalloc((void**)&d_pos_y, size);
    hipMalloc((void**)&d_pos_z, size);
    hipMalloc((void**)&d_mass, size);
    hipMalloc((void**)&d_accels, size * NUMENTITIES * 3);
    hipMalloc((void**)&d_vel_x, size);
    hipMalloc((void**)&d_vel_y, size);
    hipMalloc((void**)&d_vel_z, size);
    // Copy host memory to device memory
	hipMemcpy(d_pos_x, hPos[0], size, hipMemcpyHostToDevice);
	hipMemcpy(d_pos_y, hPos[1], size, hipMemcpyHostToDevice);
	hipMemcpy(d_pos_z, hPos[2], size, hipMemcpyHostToDevice);
	hipMemcpy(d_mass, mass, size, hipMemcpyHostToDevice);
	hipMemcpy(d_vel_x, hVel[0], size, hipMemcpyHostToDevice);
	hipMemcpy(d_vel_y, hVel[1], size, hipMemcpyHostToDevice);
	hipMemcpy(d_vel_z, hVel[2], size, hipMemcpyHostToDevice);

	// Define grid and block dimensions for pairwise acceleration computation
	dim3 blockDim(16, 16);
	dim3 gridDim((NUMENTITIES + blockDim.x - 1) / blockDim.x, (NUMENTITIES + blockDim.y - 1) / blockDim.y);

	// Compute pairwise accelerations
	pairwise_acceleration<<<gridDim, blockDim>>>(NUMENTITIES, d_pos_x, d_pos_y, d_pos_z, d_mass, d_accels);

	// Define grid dimensions for row summation
	dim3 gridDim2((NUMENTITIES + blockDim.x - 1) / blockDim.x);

	// Sum up rows to get effect on each entity, then update velocity and position
	row_summation<<<gridDim2, blockDim.x>>>(NUMENTITIES, d_accels, d_vel_x, d_vel_y, d_vel_z, d_pos_x, d_pos_y, d_pos_z);

	// Copy device memory back to host memory
	hipMemcpy(hPos[0], d_pos_x, size, hipMemcpyDeviceToHost);
	hipMemcpy(hPos[1], d_pos_y, size, hipMemcpyDeviceToHost);
	hipMemcpy(hPos[2], d_pos_z, size, hipMemcpyDeviceToHost);
	hipMemcpy(hVel[0], d_vel_x, size, hipMemcpyDeviceToHost);
	hipMemcpy(hVel[1], d_vel_y, size, hipMemcpyDeviceToHost);
	hipMemcpy(hVel[2], d_vel_z, size, hipMemcpyDeviceToHost);

	// Free device memory
	hipFree(d_pos_x);
	hipFree(d_pos_y);
	hipFree(d_pos_z);
	hipFree(d_mass);
	hipFree(d_accels);
	hipFree(d_vel_x);
	hipFree(d_vel_y);
	hipFree(d_vel_z);
}